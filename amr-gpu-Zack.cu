#include "hip/hip_runtime.h"
// defines
#define _USE_MATH_DEFINES

// cpu includes
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <cmath>
#include <unordered_map>
#include <stdexcept>
#include <chrono>

// #include <typeinfo>
// #include <typeindex>

// gpu includes
#include "cuco/static_map.cuh"
#include <thrust/device_vector.h>
#include <thrust/logical.h>
#include <thrust/transform.h>
#include <cub/block/block_reduce.cuh>
#include <cuda/std/atomic>

// namespaces
using namespace std;
using namespace std::chrono;

// constants
const int32_t LBASE = 3; // base AMR level
const int32_t LMAX = 6; // max AMR level
const int32_t NDIM = 3; // number of dimensions
const int32_t NMAX = 2097152 + 10; // maximum number of cells
const __device__ int32_t HASH[4] = {-1640531527, 97, 1003313, 5}; // hash function constants
const __device__ double FD_KERNEL[4][4] = {
    {-1., 0., 1., 3.},
    {-9., 5., 4., 15.},
    {-4., -5., 9., 15.},
    {-1., 0., 1., 2.}
};
const double rho_crit = 0.01; // critical density for refinement
const double rho_boundary = 0.; // boundary condition
const double sigma = 0.001; // std of Gaussian density field
const double EPS = 0.000001;
const string outfile_name = "grid-gpu.csv";

// custom key type
struct idx4 {
    int32_t idx3[NDIM], L;

    __host__ __device__ idx4() {}
    __host__ __device__ idx4(int32_t i_init, int32_t j_init, int32_t k_init, int32_t L_init) : idx3{i_init, j_init, k_init}, L{L_init} {}

    // Device equality operator is mandatory due to libcudacxx bug:
    // https://github.com/NVIDIA/libcudacxx/issues/223
    __device__ bool operator==(idx4 const& other) const {
        return idx3[0] == other.idx3[0] && idx3[1] == other.idx3[1] && idx3[2] == other.idx3[2] && L == other.L;
    }
};

ostream& operator<<(ostream &os, idx4 const &idx_cell) {
    os << "[" << idx_cell.idx3[0] << ", " << idx_cell.idx3[1] << ", " << idx_cell.idx3[2] << "](L=" << idx_cell.L << ")";
    return os;
}

// custom device key equal callable
struct idx4_equals {
    template <typename key_type>
    __host__ __device__ bool operator()(key_type const& lhs, key_type const& rhs) {
        return lhs.idx3[0] == rhs.idx3[0] && lhs.idx3[1] == rhs.idx3[1] && lhs.idx3[2] == rhs.idx3[2] && lhs.L == rhs.L;
    }
};

// custom value type
struct Cell {
    double rho;
    double rho_grad[3];
    int32_t flag_leaf;

    __host__ __device__ Cell() {}
    __host__ __device__ Cell(double rho_init, double rho_grad_x_init, double rho_grad_y_init, double rho_grad_z_init, 
        int32_t flag_leaf_init) : rho{rho_init}, rho_grad{rho_grad_x_init, rho_grad_y_init, rho_grad_z_init}, flag_leaf{flag_leaf_init} {}

    __host__ __device__ bool operator==(Cell const& other) const {
        return abs(rho - other.rho) < EPS && abs(rho_grad[0] - other.rho_grad[0]) < EPS
            && abs(rho_grad[1] - other.rho_grad[1]) < EPS && abs(rho_grad[2] - other.rho_grad[2]) < EPS;
    }
};

ostream& operator<<(ostream &os, Cell const &cell) {
    os << "[rho " << cell.rho << ", rho_grad_x " << cell.rho_grad[0] << ", rho_grad_y"
       << cell.rho_grad[1] << ", rho_grad_z " << cell.rho_grad[2] << ", flag_leaf " << cell.flag_leaf << "]";
    return os;
}

typedef cuco::static_map<idx4, Cell*> map_type;
typedef cuco::static_map<idx4, Cell*>::device_view map_view_type;

    // custom key type hash
    struct ramses_hash {
    template <typename key_type>
    __host__ __device__ int32_t operator()(key_type k) {
        int32_t hashval = HASH[0] * k.idx3[0] + HASH[1] * k.idx3[1] + HASH[2] * k.idx3[2] + HASH[3] * k.L;
        return hashval;
    };
};

// template<>
// struct cuco::is_bitwise_comparable<Cell> : true_type {};

// --------------- FUNCTION DECLARATIONS ------------ //
void transposeToHilbert(const int X[NDIM], const int L, int &hindex);
void hilbertToTranspose(const int hindex, const int L, int (&X)[NDIM]);
void getHindex(idx4 idx_cell, int& hindex);
void getHindexInv(int hindex, int L, idx4& idx_cell);
void makeBaseGrid(Cell (&grid)[NMAX], map_type &hashtable);
void setGridCell(const idx4 idx_cell, const int hindex, int32_t flag_leaf, map_type &hashtable);
Cell* find(map_type &hashtable, const idx4& key);
void insert(map_type &hashtable, const idx4& key, Cell* const value);
void getNeighborInfo(const idx4 idx_cell, const int dir, const bool pos, bool &is_ref, double &rho_neighbor, map_type &hashtable);
void calcGradCell(const idx4 idx_cell, Cell* cell, map_type &hashtable);
void calcGrad(map_type &hashtable);
void writeGrid(map_type &hashtable);
// zip_type retrieve_zipped(map_type &hashtable);

// ------- GLOBALS --------- //


// ------------------------------------------------ //

// convert from transposed Hilbert index to Hilbert index
void transposeToHilbert(const int X[NDIM], const int L, int &hindex) {
    int n = 0;
    hindex = 0;
    for (short i = 0; i < NDIM; ++i) {
        for (int b = 0; b < L; ++b) {
            n = (b * NDIM) + i;
            hindex |= (((X[NDIM-i-1] >> b) & 1) << n);
        }
    }
}

// convert from Hilbert index to transposed Hilbert index
void hilbertToTranspose(const int hindex, const int L, int (&X)[NDIM]) {
    int h = hindex;
    for (short i = 0; i < NDIM; ++i) X[i] = 0;
    for (short i = 0; i < NDIM * L; ++i) {
        short a = (NDIM - (i % NDIM) - 1);
        X[a] |= (h & 1) << (i / NDIM);
        h >>= 1;
    }
}

// Compute the Hilbert index for a given 4-idx (i, j, k, L)
void getHindex(idx4 idx_cell, int& hindex) {
    int X[NDIM];
    for (int i=0; i<NDIM; i++){
        X[i] = idx_cell.idx3[i];
    }
    int L = idx_cell.L;
    int m = 1 << (L - 1), p, q, t;
    int i;
    // Inverse undo
    for (q = m; q > 1; q >>= 1) {
        p = q - 1;
        for(short i = X[0]; i < NDIM; i++) {
            if (X[i] & q ) { // invert 
                X[0] ^= p;
            } else { // exchange
                t = (X[0]^X[i]) & p;
                X[0] ^= t;
                X[i] ^= t;
            }
        }
    }
    // Gray encode
    for (short i = 1; i < NDIM; i++) {
        X[i] ^= X[i-1];
    }
    t = 0;
    for (q = m; q > 1; q >>= 1) {
        if (X[NDIM - 1] & q) {
            t ^= q - 1;
        }
    }
    for (short i = 0; i < NDIM; i++) {
        X[i] ^= t;
    }
    transposeToHilbert(X, L, hindex);
}

// Compute the 3-index for a given Hilbert index and AMR level
void getHindexInv(int hindex, int L, idx4& idx_cell) {
    int X[NDIM];
    hilbertToTranspose(hindex, L, X);
    int n = 2 << (L - 1), p, q, t;
    // Gray decode by H ^ (H/2)
    t = X[NDIM - 1] >> 1;
    for (short i = NDIM - 1; i > 0; i--) {
        X[i] ^= X[i - 1];
    }
    X[0] ^= t;
    // Undo excess work
    for (q = 2; q != n; q <<= 1) {
        p = q - 1;
    }
    for (short i = NDIM - 1; i > 0; i--) {
        if(X[i] & q) { // invert
            X[0] ^= p;
        } else {
            t = (X[0]^X[i]) & p;
            X[0] ^= t;
            X[i] ^= t;
        }
    } // exchange
    for (int i=0; i<NDIM; i++) {
        idx_cell.idx3[i] = X[i];
    }
    idx_cell.L = L;
}

// globals
Cell grid[NMAX];
auto const empty_idx4_sentinel = idx4{-1, -1, -1, -1};
__host__ __device__ Cell* empty_pcell_sentinel = nullptr;

// Multi-variate Gaussian distribution
double rhoFunc(const double coord[NDIM], const double sigma) {
    double rsq = 0;
    for (short i = 0; i < NDIM; i++) {
        rsq += pow(coord[i] - 0.5, 2);
    }
    double rho = exp(-rsq / (2 * sigma)) / pow(2 * M_PI * sigma*sigma, 1.5);
    return rho;
}

// Criterion for refinement
bool refCrit(double rho) {
    return rho > rho_crit;
}

// Compute the index of the parent cell
void getParentIdx(const idx4 &idx_cell, idx4 &idx_parent) {
    for (short i = 0; i < NDIM; i++) {
        idx_parent.idx3[i] = idx_cell.idx3[i] / 2;
    }
    idx_parent.L = idx_cell.L - 1;
}

// Compute the indices of the neighbor cells in a given direction
__host__ __device__ void getNeighborIdx(const idx4 &idx_cell, const int dir, const bool pos, idx4 &idx_neighbor) {
    for (short i = 0; i < NDIM; i++) {
        idx_neighbor.idx3[i] = idx_cell.idx3[i] + (int(pos) * 2 - 1) * int(i == dir);
    }
    idx_neighbor.L = idx_cell.L;
}

// void checkIfBorder(const idx4 &idx_cell, const int dir, const bool pos, bool &is_border) {
//     is_border = idx_cell.idx3[dir] == int(pos) * (pow(2, idx_cell.L) - 1);
// }
__host__ __device__ void checkIfBorder(const idx4 &idx_cell, const int dir, const bool pos, bool &is_border) {
    is_border = idx_cell.idx3[dir] == int(pos) * (pow(2, idx_cell.L) - 1);
}

Cell* find(map_type& hashtable, const idx4& idx_cell) {
    thrust::device_vector<idx4> key;
    thrust::device_vector<Cell*> value(1);
    key.push_back(idx_cell);
    hashtable.find(key.begin(), key.end(), value.begin());
    // cout << "Searching for " << idx_cell << ", found: " << value[0] << endl;
    return value[0];
}
// GPU version: use map_view_type's find function (just one key at a time)
__device__ void find(map_view_type& hashtable, const idx4& idx_cell, Cell* pCell) {
    cuco::static_map<idx4, Cell*>::device_view::const_iterator pair = hashtable.find(idx_cell);
    // cout << "Searching for " << idx_cell << ", found: " << value[0] << endl;
    pCell = pair->second;
}

// Check if a cell exists
bool checkIfExists(const idx4& idx_cell, map_type &hashtable) {
    Cell* pCell = find(hashtable, idx_cell);
    return pCell != empty_pcell_sentinel;
}
__device__ void checkIfExists(const idx4& idx_cell, map_view_type &hashtable, bool &res) {
    Cell* pCell = nullptr;
    find(hashtable, idx_cell, pCell);
    res = pCell != empty_pcell_sentinel;
}

void makeBaseGrid(Cell (&grid)[NMAX], map_type &hashtable) {
    // not making enough leaves?
    idx4 idx_cell;
    for (int L = 0; L <= LBASE; L++) {
        for (int hindex = 0; hindex < pow(2, NDIM * L); hindex++) {
            getHindexInv(hindex, L, idx_cell);
            setGridCell(idx_cell, hindex, L == LBASE, hashtable);
        }
    }
};

void setGridCell(const idx4 idx_cell, const int hindex, int32_t flag_leaf, map_type &hashtable) {
    if (checkIfExists(idx_cell, hashtable)) throw runtime_error("setting existing cell");

    int offset;
    double dx, coord[3];
    offset = (pow(2, NDIM * idx_cell.L) - 1) / (pow(2, NDIM) - 1);

    dx = 1 / pow(2, idx_cell.L);
    for (int i = 0; i < NDIM; i++)
        coord[i] = idx_cell.idx3[i] * dx + dx / 2;
    
    grid[offset + hindex].rho = rhoFunc(coord, sigma);
    grid[offset + hindex].flag_leaf = flag_leaf;
    if (offset + hindex >= NMAX) throw runtime_error("offset () + hindex >= N_cell_max");
    insert(hashtable, idx_cell, &grid[offset + hindex]);
}

/*
*/
// TODO: this could probably also run on GPU (using a device view)
void insert(map_type &hashtable, const idx4& key, Cell* const value) {
    thrust::device_vector<idx4> insert_keys;
    thrust::device_vector<Cell*> insert_values;
    insert_keys.push_back(key);
    insert_values.push_back(value);
    // todo change this to just inserting using a pair, zip is unnecessary
    auto zipped =
        thrust::make_zip_iterator(thrust::make_tuple(insert_keys.begin(), insert_values.begin()));

    hashtable.insert(zipped, zipped + insert_keys.size());
}

void setChildrenHelper(idx4 idx_cell, short i, map_type &hashtable) {
    if (i == NDIM) {
        int hindex;
        getHindex(idx_cell, hindex);
        setGridCell(idx_cell, hindex, 1, hashtable);
        return;
    }

    setChildrenHelper(idx_cell, i+1, hashtable);
    idx_cell.idx3[i]++;
    setChildrenHelper(idx_cell, i+1, hashtable);
}


void refineGridCell(const idx4 idx_cell, map_type &hashtable) {
    int hindex;
    getHindex(idx_cell, hindex);

    Cell *pCell = find(hashtable, idx_cell);
    if (pCell == empty_pcell_sentinel) throw runtime_error("Trying to refine non-existant cell!");

    if (!pCell->flag_leaf) throw runtime_error("trying to refine non-leaf");
    if (idx_cell.L == LMAX) throw runtime_error("trying to refine at max level");
    
    // make this cell a non-leaf
    pCell->flag_leaf = 0;

    idx4 idx_child = idx_cell;
    idx_child.L++;
    for (short dir = 0; dir < NDIM; dir++) idx_child.idx3[dir] *= 2;

    // and create 2^NDIM leaf children
    setChildrenHelper(idx_child, 0, hashtable);

    // refine neighbors if needed
    idx4 idx_neighbor, idx_parent;
    int hindex_neighbor;
    for (short dir = 0; dir < NDIM; dir++) {
        for (short pos = 0; pos < 2; pos++) {
            bool is_border;
            checkIfBorder(idx_cell, dir, pos, is_border);
            if (is_border) continue;
            getNeighborIdx(idx_cell, dir, pos, idx_neighbor);
            // don't need to remove 'if' statements because this is part not for GPU (only gradient is)
            // don't need to refine if exists
            if (checkIfExists(idx_neighbor, hashtable)) continue;

            // if not exists, drop L by differen
            // we assume that L is at most different by 1
            getParentIdx(idx_cell, idx_parent);
            getNeighborIdx(idx_parent, dir, pos, idx_neighbor);
            refineGridCell(idx_neighbor, hashtable);
        }
    }
}

// zip_type retrieve_zipped(map_type& hashtable) {
//     size_t numCells = hashtable.get_size();
//     thrust::device_vector<idx4> retrieved_keys(numCells);
//     thrust::device_vector<Cell*> retrieved_values(numCells);
//     hashtable.retrieve_all(retrieved_keys.begin(), retrieved_values.begin());               // doesn't populate values for some reason
//     hashtable.find(retrieved_keys.begin(), retrieved_keys.end(), retrieved_values.begin()); // this will populate values
//     zip_type zipped = 
//         thrust::make_zip_iterator(thrust::make_tuple(retrieved_keys.begin(), retrieved_values.begin()));

//     // // Use typeid and type_index to get type information
//     // const std::type_info& typeInfo = typeid(zipped);
//     // std::type_index typeIndex = std::type_index(typeInfo);
//     // cout << "zip type: " << typeIndex.name() << endl;

//     return zipped;
// }

void refineGrid1lvl(map_type& hashtable) {
    size_t numCells = hashtable.get_size();
    thrust::device_vector<idx4> retrieved_keys(numCells);
    thrust::device_vector<Cell*> retrieved_values(numCells);
    hashtable.retrieve_all(retrieved_keys.begin(), retrieved_values.begin());               // doesn't populate values for some reason
    hashtable.find(retrieved_keys.begin(), retrieved_keys.end(), retrieved_values.begin()); // this will populate values
    auto zipped =
        thrust::make_zip_iterator(thrust::make_tuple(retrieved_keys.begin(), retrieved_values.begin()));
    // copy to an actual copy of the keys, that won't change as we refine
    thrust::device_vector<thrust::tuple<idx4, Cell*>> entries(hashtable.get_size());
    for (auto it = zipped; it != zipped + hashtable.get_size(); it++) {
        entries[it - zipped] = *it;
    }
    idx4 idx_cell;
    Cell* pCell = nullptr;
    for (auto entry : entries) { // entry is on device
        thrust::tuple<idx4, Cell*> t = entry; // t is on host
        idx_cell = t.get<0>();
        pCell = t.get<1>();
        if (refCrit(pCell->rho) && pCell->flag_leaf) {
            refineGridCell(idx_cell, hashtable); // refinement step is failing
        }
    }
}

// get information about the neighbor cell necessary for computing the gradient
void getNeighborInfo(const idx4 idx_cell, const int dir, const bool pos, bool &is_ref, double &rho_neighbor, map_type &hashtable) {
    idx4 idx_neighbor;
    int idx1_parent_neighbor;
    bool is_border, is_notref;
    // check if the cell is a border cell
    checkIfBorder(idx_cell, dir, pos, is_border);
    // compute the index of the neighbor on the same level
    getNeighborIdx(idx_cell, dir, pos, idx_neighbor);
    // if the neighbor on the same level does not exist and the cell is not a border cell, then the neighbor is not refined
    is_notref = !checkIfExists(idx_neighbor, hashtable) && !is_border;
    is_ref = !is_notref && !is_border;
    // if the cell is a border cell, set the neighbor index to the cell index (we just want a valid key for the hashtable)
    // if the neighbor is not refined, set the neighbor index to the index of the parent cell's neighbor
    // if the neighbor is refined, don't change the neighbor index
    for (short i = 0; i < NDIM; i++) {
        idx1_parent_neighbor = idx_cell.idx3[i] / 2 + (int(pos) * 2 - 1) * int(i == dir);
        idx_neighbor.idx3[i] = idx_cell.idx3[i] * int(is_border) + idx_neighbor.idx3[i] * int(is_ref) + idx1_parent_neighbor * int(is_notref);
    }
    // subtract one from the AMR level if the neighbor is not refined
    idx_neighbor.L = idx_cell.L - int(is_notref);
    // if the cell is a border cell, use the boundary condition
    Cell* pCell = find(hashtable, idx_neighbor);
    rho_neighbor = pCell->rho * int(!is_border) + rho_boundary * int(is_border);
}
// GPU VERISON: get information about the neighbor cell necessary for computing the gradient
__device__ void getNeighborInfo(const idx4 idx_cell, const int dir, const bool pos, bool &is_ref, double &rho_neighbor, map_view_type &hashtable) {
    idx4 idx_neighbor;
    int idx1_parent_neighbor;
    bool is_border, is_notref, exists;
    // check if the cell is a border cell
    checkIfBorder(idx_cell, dir, pos, is_border);
    // compute the index of the neighbor on the same level
    getNeighborIdx(idx_cell, dir, pos, idx_neighbor);
    // if the neighbor on the same level does not exist and the cell is not a border cell, then the neighbor is not refined
    checkIfExists(idx_neighbor, hashtable, exists); 
    is_notref = !exists && !is_border;
    is_ref = !is_notref && !is_border;
    // if the cell is a border cell, set the neighbor index to the cell index (we just want a valid key for the hashtable)
    // if the neighbor is not refined, set the neighbor index to the index of the parent cell's neighbor
    // if the neighbor is refined, don't change the neighbor index
    for (short i = 0; i < NDIM; i++) {
        idx1_parent_neighbor = idx_cell.idx3[i] / 2 + (int(pos) * 2 - 1) * int(i == dir);
        idx_neighbor.idx3[i] = idx_cell.idx3[i] * int(is_border) + idx_neighbor.idx3[i] * int(is_ref) + idx1_parent_neighbor * int(is_notref);
    }
    // subtract one from the AMR level if the neighbor is not refined
    idx_neighbor.L = idx_cell.L - int(is_notref);
    // if the cell is a border cell, use the boundary condition
    Cell* pCell = nullptr;
    find(hashtable, idx_neighbor, pCell);
    rho_neighbor = pCell->rho * int(!is_border) + rho_boundary * int(is_border);
}

// compute the gradient for one cell
__device__ void calcGradCell(const idx4 idx_cell, Cell* cell, map_view_type &hashtable) {
    bool is_ref[2];
    double dx, rho[3];
    int fd_case;
    dx = pow(0.5, idx_cell.L);
    rho[2] = cell->rho;
    for (short dir = 0; dir < NDIM; dir++) {
        for (short pos = 0; pos < 2; pos++) {
            getNeighborInfo(idx_cell, dir, pos, is_ref[pos], rho[pos], hashtable);
        }
        fd_case = is_ref[0] + 2 * is_ref[1];
        cell->rho_grad[dir] = (FD_KERNEL[fd_case][0] * rho[0] + FD_KERNEL[fd_case][1] * rho[2] + FD_KERNEL[fd_case][2] * rho[1]) / (FD_KERNEL[fd_case][3] * dx);
    }
}

// compute the gradient
__global__ void calcGrad(map_view_type &hashtable, auto zipped, size_t hashtable_size) {
    idx4 idx_cell;
    Cell* pCell = nullptr;
    for (auto it = zipped; it != zipped + hashtable_size; it++) {
        thrust::tuple<idx4, Cell*> t = *it;
        idx_cell = t.get<0>();
        pCell = t.get<1>();
        calcGradCell(idx_cell, pCell, hashtable);
    }
}

void writeGrid(map_type& hashtable) {
    // save i, j, k, L, rho, gradients for all cells (use the iterator) to a file
    ofstream outfile;
    outfile.open(outfile_name);
    idx4 idx_cell;
    Cell* pCell = nullptr;
    outfile << "i,j,k,L,flag_leaf,rho,rho_grad_x,rho_grad_y,rho_grad_z\n";
    size_t numCells = hashtable.get_size();
    thrust::device_vector<idx4> retrieved_keys(numCells);
    thrust::device_vector<Cell*> retrieved_values(numCells);
    hashtable.retrieve_all(retrieved_keys.begin(), retrieved_values.begin());               // doesn't populate values for some reason
    hashtable.find(retrieved_keys.begin(), retrieved_keys.end(), retrieved_values.begin()); // this will populate values
    auto zipped =
        thrust::make_zip_iterator(thrust::make_tuple(retrieved_keys.begin(), retrieved_values.begin()));
    for (auto it = zipped; it != zipped + hashtable.get_size(); it++) {
        thrust::tuple<idx4, Cell*> t = *it;
        idx_cell = t.get<0>();
        pCell = t.get<1>();
        outfile << idx_cell.idx3[0] << "," << idx_cell.idx3[1] << "," << idx_cell.idx3[2]
                << "," << idx_cell.L << "," << pCell->flag_leaf << "," << pCell->rho << "," << pCell->rho_grad[0]
                << "," << pCell->rho_grad[1] << "," << pCell->rho_grad[2] << "\n";
    }
    outfile.close();
}



// all tests (later move out)

void test_full_output() {
    // Cell empty_cell_sentinel{-1, -1, -1, -1, -1};
    cuco::static_map<idx4, Cell*> hashtable{
        NMAX, cuco::empty_key{empty_idx4_sentinel}, cuco::empty_value{empty_pcell_sentinel}
    };

    // grid memory accessible from CPU or GPU?
    // hipMallocManaged(&x, N * sizeof(float));

    cout << "Making base grid" << endl;
    makeBaseGrid(grid, hashtable);
    const int num_ref = LMAX - LBASE;
    cout << "Refining grid levels" << endl;
    for (short i = 0; i < num_ref; i++) {
       refineGrid1lvl(hashtable);
    }
    cout << "Finished refining grid levels" << endl;

    cout << "Calculating gradients" << endl;
    auto start = high_resolution_clock::now();

    // run as kernel on GPU
    map_view_type view = hashtable.get_device_view();
    // get zipped values before kicking off kernels
    size_t numCells = hashtable.get_size();
    thrust::device_vector<idx4> retrieved_keys(numCells);
    thrust::device_vector<Cell*> retrieved_values(numCells);
    hashtable.retrieve_all(retrieved_keys.begin(), retrieved_values.begin());               // doesn't populate values for some reason
    hashtable.find(retrieved_keys.begin(), retrieved_keys.end(), retrieved_values.begin()); // this will populate values
    auto zipped =
        thrust::make_zip_iterator(thrust::make_tuple(retrieved_keys.begin(), retrieved_values.begin()));
    calcGrad<<<8, 2>>>(view, zipped, hashtable.get_size());

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);
    cout << duration.count() << " ms" << endl;
    writeGrid(hashtable);
}

void test_map_insert_int() {
    cuco::static_map<int32_t, int32_t> hashtable{NMAX, cuco::empty_key{-1}, cuco::empty_value{-1}};
    thrust::device_vector<cuco::pair<int32_t, int32_t>> test_pair;
    test_pair.push_back(pair<int32_t, int32_t>(2, 3));
    hashtable.insert(test_pair.begin(), test_pair.end()); // was not working

    // Retrieve contents of all the non-empty slots in the map
    thrust::device_vector<int32_t> result_keys(2);
    thrust::device_vector<int32_t> result_values(2);
    hashtable.retrieve_all(result_keys.begin(), result_values.begin());

    cout << "KEYS:" << endl;
    for (auto k : result_keys) {
        cout << k << endl;
    }

    cout << "VALUES:" << endl;
    for (auto v : result_values) {
        cout << v << endl;
    }
}

void test_map_insert_cell_pointer() {
    cuco::static_map<idx4, Cell*> hashtable{
        NMAX, cuco::empty_key{empty_idx4_sentinel}, cuco::empty_value{empty_pcell_sentinel}
    };

    // Retrieve contents of all the non-empty slots in the map
    thrust::device_vector<idx4> result_keys(2);
    thrust::device_vector<Cell*> result_values(2);
    // hashtable.find(insert_keys.begin(), insert_keys.end(), result_values.begin());
    // hashtable.retrieve_all(result_keys.begin(), result_values.begin());

    // cout << "KEYS:" << endl;
    // for (auto k : result_keys) {
    //     cout << k << endl;
    // }

    Cell* pResult;
    cout << "VALUES:" << endl;
    for (auto v : result_values) {
        // v is a pointer to Cell
        pResult = v;
        cout << pResult << endl;
    }

    //delete pTest_cell;
}
 
void test_map_insert_cell_pointer_Roma() {
    idx4 idx_cell{1, 1, 1, 1};
    Cell* pTest_cell = new Cell{1, 1, 1, 1, 1}; // create on heap
    cuco::static_map<idx4, Cell*> hashtable{
        NMAX, cuco::empty_key{empty_idx4_sentinel}, cuco::empty_value{empty_pcell_sentinel}
    };

    cout << "address of test_cell:" << pTest_cell << endl;
    cout << "test cell rho:" << pTest_cell->rho << endl;

    thrust::device_vector<idx4> insert_keys;
    insert_keys.push_back(idx_cell);
    thrust::device_vector<Cell*> insert_values;
    insert_values.push_back(pTest_cell);
    auto zipped =
        thrust::make_zip_iterator(thrust::make_tuple(insert_keys.begin(), insert_values.begin()));

    // trying zip iterator
    hashtable.insert(zipped, zipped + insert_keys.size());

    bool test_exist;
    test_exist = checkIfExists(idx_cell, hashtable);
    cout << "KEY EXISTS? " << test_exist << endl;
    test_exist = checkIfExists(idx4{1,2,3,4}, hashtable);
    cout << "FAKE KEY EXISTS? " << test_exist << endl;

    // trying retrieve all
    // Retrieve contents of all the non-empty slots in the map
    // why retrieve all not working?
    thrust::device_vector<idx4> result_keys(2);
    thrust::device_vector<Cell*> result_values(2);
    hashtable.retrieve_all(result_keys.begin(), result_values.begin());
    hashtable.find(result_keys.begin(), result_keys.end(), result_values.begin());

    cout << "KEYS:" << endl;
    for (auto k : result_keys) {
        cout << k << endl;
    }

    cout << "VALUES:" << endl;
    for (auto v : result_values) {
        cout << v << endl;
    }
}

void test_map_insert_cell_pointer_Roma2() {
    using Key = idx4;
    using Value = Cell*;
    // cuco::static_map<Key, Value> hashtable{NMAX, cuco::empty_key{-1}, cuco::empty_value{-1}};

    idx4 idx_cell{1, 1, 1, 1};
    Cell* pTest_cell = new Cell{1, 1, 1, 1, 1}; // create on heap
    cout << "Address of test cell: " << pTest_cell << endl;
    cuco::static_map<Key, Value> hashtable{
        NMAX, cuco::empty_key{empty_idx4_sentinel}, cuco::empty_value{empty_pcell_sentinel}
    };

    thrust::device_vector<cuco::pair<Key, Value>> test_pair;
    // test_pair.push_back(pair<Key, Value>(2, 3));
    test_pair.push_back(pair<Key, Value>(idx_cell, pTest_cell));
    hashtable.insert(test_pair.begin(), test_pair.end());

    // check if its in the table correctly?
    Cell* pResult;
    pResult = find(hashtable, idx_cell);
    cout << "Found" << pResult << endl;
    cout << "Size: " << hashtable.get_size() << endl;

    // Retrieve contents of all the non-empty slots in the map
    thrust::device_vector<Key> result_keys(1);
    thrust::device_vector<Value> result_values(1);
    // roundabout solution since retrieve_all not getting values: use it to get all keys, and then pass into find to get all values
    hashtable.retrieve_all(result_keys.begin(), result_values.begin());
    hashtable.find(result_keys.begin(), result_keys.end(), result_values.begin());

    cout << "KEYS:" << endl;
    for (auto k : result_keys) {
        cout << k << endl;
    }

    cout << "VALUES:" << endl;
    for (auto v : result_values) {
        cout << v << endl;
    }
}

void testHilbert() {
    idx4 idx_cell, idx_cell2;
    cin >> idx_cell.idx3[0];
    cin >> idx_cell.idx3[1];
    cin >> idx_cell.idx3[2];
    idx_cell.L = 2;
    int hindex;
    getHindex(idx_cell, hindex);
    cout << hindex << endl;
    // test inverse
    getHindexInv(hindex, 2, idx_cell2);
    cout << "Inverse of hindex=" << hindex << " is " << idx_cell2 << endl;
}

int main() {
    test_full_output();
}
